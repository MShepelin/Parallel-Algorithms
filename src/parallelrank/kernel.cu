#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "CUDA-By-Example/common/book.h"
#include "CUDA-By-Example/common/cpu_bitmap.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define GRID_DIM 1000

#define RANK_SEARCH_FLAGS_SIZE 1

#define PAIRS_PER_ROUND 65536
#define BLOCKS_FOR_PAIRS_SEARCH 256
#define INVALID_PAIR_VALUE -1

#define cudaCheckError(msg) {  \
	hipError_t __err = hipGetLastError();  \
	if(__err != hipSuccess) {  \
		fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					(msg), hipGetErrorString(__err), \
					__FILE__, __LINE__); \
		fprintf(stderr, "*** FAILED - ABORTING\n"); \
		exit(1); \
	} \
}

__global__ void find_subtraction_pairs_raw(int32_t* nnz_estimation, int32_t* subtraction_pairs, int32_t* d_column_sizes, uint32_t columns) {
	// Assumes subtraction_pairs has size (PAIRS_PER_ROUND * 2)
	
	// Each block has N threads
	// Each thread works for a unique column and 
	// checks all columns with lower indexes (starting from left)

	// Assumes memory_calculation has size gridDim.x

	__shared__ int32_t new_subtraction_id;

	if (threadIdx.x == 0) {
		new_subtraction_id = 0;
	}

	__syncthreads();
	
	uint32_t max_subtractions = PAIRS_PER_ROUND / gridDim.x;
	uint32_t offset = blockIdx.x * max_subtractions;
	// TODO: add assertion that PAIRS_PER_ROUND % gridDim.x == 0
	// TODO: may be make them static

	for (size_t column_id = blockIdx.x * blockDim.x + threadIdx.x; column_id < columns; column_id += gridDim.x * blockDim.x) {
		bool is_subtraction_found = false;
		for (size_t left_column_id = 0; left_column_id < columns; ++left_column_id) {
			if (d_column_sizes[column_id] == d_column_sizes[left_column_id]) {
				int32_t old_new_subtraction_id = atomicAdd(&new_subtraction_id, 1);
				if (old_new_subtraction_id >= max_subtractions) {
					// Block batch is full
					break;
				}

				is_subtraction_found = true;
				nnz_estimation[column_id] = d_column_sizes[column_id] + d_column_sizes[left_column_id] - 2;
				subtraction_pairs[(offset + old_new_subtraction_id) * 2] = column_id;
				subtraction_pairs[(offset + old_new_subtraction_id) * 2 + 1] = left_column_id;
				// subtraction pair means columns[column_id] -= columns[left_column_id]
				break;
			}
		}

		if (!is_subtraction_found) {
			// No atomic operations are needed because 
			// each column_id is devoted to one thread
			nnz_estimation[column_id] = d_column_sizes[column_id];
		}
	}
}

__global__ void check_if_matrix_reduced_raw(
	int32_t* rank_search_flags,
	int32_t* d_column_sizes, 
	uint32_t columns) {
	// Check every pair of (i, j) where i and j are column indicies
	size_t columns_pairs = columns * columns;
	for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < columns_pairs; i += gridDim.x * blockDim.x) {
		size_t column_left = i % columns;
		size_t column_right = i - column_left * columns;
		if (d_column_sizes[column_left] == d_column_sizes[column_right]) {
			atomicOr(rank_search_flags, 1);
		}
	}
}

__global__ void fill_column_sizes(int32_t* d_column_sizes, uint32_t columns, int32_t* d_columns_offsets) {
	// Assumes d_columns_offsets has size of (columns + 1)
	for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < columns; i += gridDim.x * blockDim.x) {
		d_column_sizes[i] = d_columns_offsets[i + 1] - d_columns_offsets[i];
	}
}

struct CSRMatrix {
public:
	thrust::device_vector<int32_t> d_columns_offsets;
	thrust::device_vector<int32_t> d_rows_indicies;
	// Number of real elements in column,
	// is <= (difference in d_columns_offsets neighbour elements)
	thrust::device_vector<int32_t> d_column_sizes; 

public:
	CSRMatrix() = delete;

	CSRMatrix(int32_t columns) {
		d_column_sizes.assign(columns, 0);
		d_columns_offsets.assign(columns + 1, -1);
		// We put invalid size value
		// TODO: check that d_columns_offsets really has size (columns + 1)
	}

	CSRMatrix(int32_t* column_offsets, uint32_t column_offsets_len, int32_t* rows_indicies, uint32_t nnz, int32_t columns) {
		d_columns_offsets.assign(column_offsets, column_offsets + column_offsets_len);
		d_rows_indicies.assign(rows_indicies, rows_indicies + nnz);
		d_column_sizes.assign(columns, 0);
		fill_column_sizes<<<256, 256>>>(
			thrust::raw_pointer_cast(d_column_sizes.data()), d_column_sizes.size(),
			thrust::raw_pointer_cast(d_columns_offsets.data())); // TODO: fix grid size
	}

	void check_if_matrix_reduced(thrust::device_vector<int32_t>& rank_search_flags) {
		check_if_matrix_reduced_raw<<<256, 256>>>( // TODO: fix grid size
			thrust::raw_pointer_cast(rank_search_flags.data()),
			thrust::raw_pointer_cast(d_column_sizes.data()),
			d_column_sizes.size());
	}

	void find_subtraction_pairs(
		thrust::device_vector<int32_t>& d_nnz_estimation,
		thrust::device_vector<int32_t>& d_pairs_for_subtractions) {
		find_subtraction_pairs_raw<<<BLOCKS_FOR_PAIRS_SEARCH, 256>>>(
			thrust::raw_pointer_cast(d_nnz_estimation.data()),
			thrust::raw_pointer_cast(d_pairs_for_subtractions.data()),
			thrust::raw_pointer_cast(d_column_sizes.data()),
			d_column_sizes.size()
		);
	}

	// TODO: add squash method to remove all garbage data in d_rows_indicies

	//void prepare_memory(uint32_t nnz) {
		// d_column_sizes
	//}

	void update_columns_offsets(thrust::device_vector<int32_t>& d_nnz_estimation) {
		// TODO: figure out a better way to update columns offsets
		thrust::host_vector<int32_t> nnz_estimation = d_nnz_estimation;
		thrust::host_vector<int32_t> new_columns_offsets;
		new_columns_offsets.assign(d_column_sizes.size() + 1, 0);

		for (size_t i = 1; i < d_column_sizes.size() + 1; ++i) {
			new_columns_offsets[i] = new_columns_offsets[i - 1] + nnz_estimation[i - 1];
		}

		d_columns_offsets = new_columns_offsets;
	}
};

extern "C" void read_CSR(int32_t* column_offsets, uint32_t column_offsets_len, int32_t* rows_indicies, uint32_t nnz, int32_t columns, int32_t rows) {
	CSRMatrix buffers[] = {
		CSRMatrix(column_offsets, column_offsets_len, rows_indicies, nnz, columns),
		CSRMatrix(columns)
	};
	uint32_t active_buffer_index = 0;

	
	thrust::device_vector<int32_t> rank_search_flags(RANK_SEARCH_FLAGS_SIZE, false);
	// Structure of rank_search_flags:
	// 0) is matrix reduced?

	thrust::device_vector<int32_t> d_pairs_for_subtractions(PAIRS_PER_ROUND * 2, -1);
	thrust::device_vector<int32_t> d_nnz_estimation(columns, 0);

	cudaCheckError("Buffer initialisation");

	// Do while not reduced:
	for (int32_t attempt = 0; (attempt < 1) && (!rank_search_flags[0]); ++attempt) {
		// TODO: figure out a better way to check boolean
		// TODO: define maimum attempts or take it from function arguements
		d_pairs_for_subtractions.assign(PAIRS_PER_ROUND * 2, INVALID_PAIR_VALUE);
		buffers[active_buffer_index].find_subtraction_pairs(d_nnz_estimation, d_pairs_for_subtractions);
		// TODO: check that values (from) don't repeat in pairs value
		// TODO: check that all columns are set in d_nnz_estimation
		buffers[active_buffer_index].update_columns_offsets(d_nnz_estimation);
		// perform subtraction with merge
		// ???

		active_buffer_index = 1 - active_buffer_index;
		buffers[active_buffer_index].check_if_matrix_reduced(rank_search_flags);
		cudaCheckError("Matrix reduction check");

		// [IMPORTANT] check that algorithm works when -1 can be found in rows_indicies (extra memory space) and column_size (empty columns)
	}
}
