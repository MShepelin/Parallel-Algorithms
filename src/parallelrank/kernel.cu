#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "CUDA-By-Example/common/book.h"
#include "CUDA-By-Example/common/cpu_bitmap.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <stdint.h>

#define GRID_DIM 1000

#define RANK_SEARCH_FLAGS_SIZE 1

#define PAIRS_PER_ROUND 65536
#define BLOCKS_FOR_PAIRS_SEARCH 256
#define INVALID_PAIR_VALUE -1

#define cudaCheckError(msg) {  \
	hipError_t __err = hipGetLastError();  \
	if(__err != hipSuccess) {  \
		fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					(msg), hipGetErrorString(__err), \
					__FILE__, __LINE__); \
		fprintf(stderr, "*** FAILED - ABORTING\n"); \
		exit(1); \
	} \
}

// TODO: change type in subtraction_pairs for uint32_t
__global__ void perform_subtractions(
	const int32_t* subtraction_pairs,
	const int32_t* input_columns_offsets,
	const int32_t* input_column_sizes,
	const int32_t* input_rows_indicies,
	const int32_t* output_columns_offsets,
	int32_t* output_column_sizes,
	int32_t* output_rows_indicies) {
	// Assumes subtraction_pairs has size (PAIRS_PER_ROUND * 2)
	for (int32_t pair_id = blockIdx.x * blockDim.x + threadIdx.x; pair_id < PAIRS_PER_ROUND; pair_id += gridDim.x * blockDim.x) {
		int32_t column_from = subtraction_pairs[pair_id * 2];
		int32_t column_subtraction = subtraction_pairs[pair_id * 2 + 1];

		uint32_t id_to_put = output_columns_offsets[column_from];
		uint32_t left_column_id = input_columns_offsets[column_from];
		const uint32_t left_column_id_limit = left_column_id + input_column_sizes[column_from];
		uint32_t right_column_id = input_columns_offsets[column_subtraction];
		const uint32_t right_column_id_limit = right_column_id + input_column_sizes[column_subtraction];

		while (left_column_id < left_column_id_limit ||
			right_column_id < right_column_id_limit) {
			
			uint32_t left_low = (left_column_id < left_column_id_limit) ? input_rows_indicies[left_column_id] : UINT32_MAX;
			uint32_t right_low = (right_column_id < right_column_id_limit) ? input_rows_indicies[right_column_id] : UINT32_MAX;

			if (left_low == right_low) {
				++left_column_id;
				++right_column_id;
				// 1 ^ 1 = 0
			}
			else if (left_low < right_low) {
				output_rows_indicies[id_to_put] = left_low;
				++output_column_sizes[column_from];
				++id_to_put;
				++left_column_id;
				// 1 ^ 0 = 1
			}
			else if (right_low < left_low) {
				output_rows_indicies[id_to_put] = right_low;
				++output_column_sizes[column_from];
				++id_to_put;
				++right_column_id;
				// 0 ^ 1 = 1
			}
		}
	}
}

__global__ void find_subtraction_pairs_raw(int32_t* nnz_estimation, int32_t* subtraction_pairs, int32_t* column_sizes, uint32_t columns) {
	// Assumes subtraction_pairs has size (PAIRS_PER_ROUND * 2)
	
	// Each block has N threads
	// Each thread works for a unique column and 
	// checks all columns with lower indexes (starting from left)

	// Assumes memory_calculation has size gridDim.x

	__shared__ int32_t new_subtraction_id;

	if (threadIdx.x == 0) {
		new_subtraction_id = 0;
	}

	__syncthreads();
	
	uint32_t max_subtractions = PAIRS_PER_ROUND / gridDim.x;
	uint32_t offset = blockIdx.x * max_subtractions;
	// TODO: add assertion that PAIRS_PER_ROUND % gridDim.x == 0
	// TODO: may be make them static

	for (size_t column_id = blockIdx.x * blockDim.x + threadIdx.x; column_id < columns; column_id += gridDim.x * blockDim.x) {
		bool is_subtraction_found = false;

		if (column_sizes[column_id] > 0) {
			for (size_t left_column_id = 0; left_column_id < column_id; ++left_column_id) {
				if (column_sizes[column_id] == column_sizes[left_column_id]) {
					int32_t old_new_subtraction_id = atomicAdd(&new_subtraction_id, 1);
					if (old_new_subtraction_id >= max_subtractions) {
						// Block batch is full
						break;
					}

					is_subtraction_found = true;
					nnz_estimation[column_id] = column_sizes[column_id] + column_sizes[left_column_id] - 2;
					subtraction_pairs[(offset + old_new_subtraction_id) * 2] = column_id;
					subtraction_pairs[(offset + old_new_subtraction_id) * 2 + 1] = left_column_id;
					// subtraction pair means columns[column_id] -= columns[left_column_id]
					break;
				}
			}
		}

		if (!is_subtraction_found) {
			// No atomic operations are needed because 
			// each column_id is devoted to one thread
			nnz_estimation[column_id] = column_sizes[column_id];
		}
	}
}

__global__ void check_if_matrix_reduced_raw(
	int32_t* rank_search_flags,
	int32_t* column_sizes, 
	uint32_t columns) {
	// Check every pair of (i, j) where i and j are column indicies
	size_t columns_pairs = columns * columns;
	for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < columns_pairs; i += gridDim.x * blockDim.x) {
		size_t column_left = i % columns;
		size_t column_right = i - column_left * columns;
		if (column_sizes[column_left] == column_sizes[column_right]) {
			atomicOr(rank_search_flags, 1);
		}
	}
}

__global__ void fill_column_sizes(int32_t* column_sizes, uint32_t columns, int32_t* columns_offsets) {
	// Assumes columns_offsets has size of (columns + 1)
	for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < columns; i += gridDim.x * blockDim.x) {
		column_sizes[i] = columns_offsets[i + 1] - columns_offsets[i];
	}
}

struct CSRMatrix {
public:
	thrust::device_vector<int32_t> d_columns_offsets;
	thrust::device_vector<int32_t> d_rows_indicies;
	// Number of real elements in column,
	// is <= (difference in d_columns_offsets neighbour elements)
	thrust::device_vector<int32_t> d_column_sizes; 

public:
	CSRMatrix() = delete;

	CSRMatrix(int32_t columns) {
		d_column_sizes.assign(columns, 0);
		d_columns_offsets.assign(columns + 1, -1);
		// We put invalid size value
		// TODO: check that d_columns_offsets really has size (columns + 1)
	}

	CSRMatrix(int32_t* column_offsets, uint32_t column_offsets_len, int32_t* rows_indicies, uint32_t nnz, int32_t columns) {
		d_columns_offsets.assign(column_offsets, column_offsets + column_offsets_len);
		d_rows_indicies.assign(rows_indicies, rows_indicies + nnz);
		d_column_sizes.assign(columns, 0);
		fill_column_sizes<<<256, 256>>>(
			thrust::raw_pointer_cast(d_column_sizes.data()), d_column_sizes.size(),
			thrust::raw_pointer_cast(d_columns_offsets.data())); // TODO: fix grid size
	}

	void check_if_matrix_reduced(thrust::device_vector<int32_t>& rank_search_flags) {
		check_if_matrix_reduced_raw<<<256, 256>>>( // TODO: fix grid size
			thrust::raw_pointer_cast(rank_search_flags.data()),
			thrust::raw_pointer_cast(d_column_sizes.data()),
			d_column_sizes.size());
	}

	void find_subtraction_pairs(
		thrust::device_vector<int32_t>& d_nnz_estimation,
		thrust::device_vector<int32_t>& d_pairs_for_subtractions) {
		find_subtraction_pairs_raw<<<BLOCKS_FOR_PAIRS_SEARCH, 256>>>(
			thrust::raw_pointer_cast(d_nnz_estimation.data()),
			thrust::raw_pointer_cast(d_pairs_for_subtractions.data()),
			thrust::raw_pointer_cast(d_column_sizes.data()),
			d_column_sizes.size()
		);
	}

	// TODO: add squash method to remove all garbage data in d_rows_indicies

	void perform_subtraction(CSRMatrix& output, const thrust::device_vector<int32_t>& d_pairs_for_subtractions) const {
		perform_subtractions<<<256, 256>>>(
			thrust::raw_pointer_cast(d_pairs_for_subtractions.data()),
			thrust::raw_pointer_cast(d_columns_offsets.data()),
			thrust::raw_pointer_cast(d_column_sizes.data()),
			thrust::raw_pointer_cast(d_rows_indicies.data()),

			thrust::raw_pointer_cast(output.d_columns_offsets.data()),
			thrust::raw_pointer_cast(output.d_column_sizes.data()),
			thrust::raw_pointer_cast(output.d_rows_indicies.data())
		);
	}

	void update_columns_offsets(thrust::device_vector<int32_t>& d_nnz_estimation) {
		// TODO: figure out a better way to update columns offsets
		thrust::host_vector<int32_t> nnz_estimation = d_nnz_estimation;
		thrust::host_vector<int32_t> new_columns_offsets;
		new_columns_offsets.assign(d_column_sizes.size() + 1, 0);

		for (size_t i = 1; i < d_column_sizes.size() + 1; ++i) {
			new_columns_offsets[i] = new_columns_offsets[i - 1] + nnz_estimation[i - 1];
		}

		d_columns_offsets = new_columns_offsets;
		d_rows_indicies.assign(new_columns_offsets[d_column_sizes.size()], -1);
		d_column_sizes.assign(d_column_sizes.size(), 0);
	}
};

extern "C" void read_CSR(int32_t* column_offsets, uint32_t column_offsets_len, int32_t* rows_indicies, uint32_t nnz, int32_t columns, int32_t rows) {
	CSRMatrix buffers[] = {
		CSRMatrix(column_offsets, column_offsets_len, rows_indicies, nnz, columns),
		CSRMatrix(columns)
	};
	uint32_t active_buffer_index = 0;
	
	thrust::device_vector<int32_t> rank_search_flags(RANK_SEARCH_FLAGS_SIZE, false);
	// Structure of rank_search_flags:
	// 0) is matrix reduced?

	thrust::device_vector<int32_t> d_pairs_for_subtractions(PAIRS_PER_ROUND * 2, -1);
	thrust::device_vector<int32_t> d_nnz_estimation(columns, 0);

	cudaCheckError("Buffer initialisation");

	// Do while not reduced:
	for (int32_t attempt = 0; (attempt < 1) && (!rank_search_flags[0]); ++attempt) {
		// TODO: figure out a better way to check boolean
		// TODO: define maimum attempts or take it from function arguements
		d_pairs_for_subtractions.assign(PAIRS_PER_ROUND * 2, INVALID_PAIR_VALUE);
		buffers[active_buffer_index].find_subtraction_pairs(d_nnz_estimation, d_pairs_for_subtractions);
		// TODO: check that values (from) don't repeat in pairs value
		// TODO: check that all columns are set in d_nnz_estimation
		buffers[1 - active_buffer_index].update_columns_offsets(d_nnz_estimation);
		// perform subtraction with merge
		buffers[active_buffer_index].perform_subtraction(buffers[1 - active_buffer_index], d_pairs_for_subtractions);

		active_buffer_index = 1 - active_buffer_index;
		buffers[active_buffer_index].check_if_matrix_reduced(rank_search_flags);
		cudaCheckError("Matrix reduction check");

		// [IMPORTANT] check that algorithm works when -1 can be found in rows_indicies (extra memory space) and column_size (empty columns)
	}
}
